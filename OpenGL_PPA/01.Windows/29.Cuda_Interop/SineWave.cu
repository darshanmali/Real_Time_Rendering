
#include <hip/hip_runtime.h>

__global__ void SineWave(float4* pos, unsigned int mess_Width, unsigned int mess_Height, float time)
{
	
   unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
   unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

   float u = x / (float)mess_Width;
   float v = y / (float)mess_Height;
   u = u * 2.0f - 1.0f;
   v = v * 2.0f - 1.0f;
   float freq = 4.0f;
   float w = sinf(u * freq + time) + cosf(v * freq + time) * 1.0f;

   pos[y * mess_Width + x] = make_float4(u, w, v, 1.0f);

}

void LaunchCUDAKernal(float4* pos, unsigned int mess_Width, unsigned int mess_Height, float time)
{
	dim3 DimBlock = dim3(8, 8, 1);
	dim3 DimGrid = dim3(mess_Width/ DimBlock.x, mess_Height / DimBlock.y , 1);

	SineWave<<<DimGrid, DimBlock >>> (pos, mess_Width, mess_Height, time);

}


